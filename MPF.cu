#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <vector>
#include <iostream>
#include <lapacke.h>
#include <hipblas.h>
#include "fp16_utils.h"
#include "hgetf2_kernel.h"
#include "dgetf2_native_npv.h"
#include "cuda_debug.h"

#define __threads_per_block__ 256

// Quick calculation of blocks needed based on the number of threads needed
int inline grid_size(int threads_needed) {
    return (threads_needed + __threads_per_block__ - 1) / __threads_per_block__;
}

// GPU kernel for FP64 to FP16 conversion
__global__ void double_to_fp16_block(const double *input, fp16 *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = double_to_fp16(input[idx]);
    }
}

// GPU kernel for FP16 to FP64 conversion
__global__ void fp16_to_double_block(const fp16 *input, double *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fp16_to_double(input[idx]);
    }
}
// GPU kernel for applying row swaps based on pivot indices
__global__ void LASWP_kernel(double *A, int lda, int k, int cols, const int *ipiv_panel) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    if (col < lda) {
        // Apply swaps sequentially for this column
        for (int panel_col = 0; panel_col < cols; ++panel_col) {
            int current_row = k + panel_col;              // Current row being processed
            int pivot_row = ipiv_panel[panel_col] - 1;    // Convert to 0-based global index

            if (pivot_row != current_row) {
                // Swap A[col * lda + current_row] <-> A[col * lda + pivot_row]
                double tmp = A[col * lda + current_row];
                A[col * lda + current_row] = A[col * lda + pivot_row];
                A[col * lda + pivot_row] = tmp;
            }
        }
    }

}


void MPF(double *A, int N, int r, int *IPIV) {
    
    // Check CUDA device availability
    int deviceCount;
    hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices available." << std::endl;
        return;
    }

    hipSetDevice(0);  // Explicitly set device

    // Allocate device memory
    double *d_A;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);

    fp16 *d_P_FP16_buffer;
    hipMalloc(&d_P_FP16_buffer, N * r * sizeof(fp16));

    double *d_P_FP64_NPV_buffer;
    hipMalloc(&d_P_FP64_NPV_buffer, N * r * sizeof(double));

    int *d_IPIV_panel;
    hipMalloc(&d_IPIV_panel, r * sizeof(int));

    int *d_IPIV;
    hipMalloc(&d_IPIV, N * sizeof(int));

    hipblasHandle_t handle;
    hipblasStatus_t hipblasStatus_t = hipblasCreate(&handle);

    // Panel iteration
    for (int k = 0; k < N; k += r) {
        int panel_cols = std::min(r, N - k); // Number of columns in the current panel (r or N%r)
        int panel_rows = N - k; // Number of rows in the panel

        if (panel_rows > 1) {

            // 1.1 Extract panel from matrix A to FP64 buffer
            // Copy panel column by column using hipMemcpy
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    d_A + (k + col) * N + k,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }
            hipDeviceSynchronize();
            std::vector<double> h_P_FP64_NPV_buffer(panel_rows * panel_cols);
            hipMemcpy(h_P_FP64_NPV_buffer.data(), d_P_FP64_NPV_buffer, panel_rows * panel_cols * sizeof(double), hipMemcpyDeviceToHost);
            // 1.2 Convert and copy FP64 panel to FP16 panel
            int total_elements = panel_rows * panel_cols;
            double_to_fp16_block << <grid_size(total_elements), __threads_per_block__ >> > (d_P_FP64_NPV_buffer, d_P_FP16_buffer, total_elements);
            hipDeviceSynchronize();



            // 2 Panel LU factorization in FP16 using Cooperative Groups
            int num_blocks = grid_size(panel_rows);
            int threads_per_block = __threads_per_block__;
            
            void* args[] = {&d_P_FP16_buffer, &panel_rows, &panel_rows, &panel_cols, &d_IPIV_panel};
            
            
            hipError_t err = hipLaunchCooperativeKernel((void*)HGETF2_kernel, 
                                                        dim3(num_blocks), dim3(threads_per_block), 
                                                        args, 0, 0);
            if (err != hipSuccess) {
                std::cout << "CUDA HGETF2 kernel error: " << hipGetErrorString(err) 
                         << " (panel_rows=" << panel_rows << ", num_blocks=" << num_blocks 
                         << ", threads_per_block=" << threads_per_block << ")" << std::endl;
                return;
            }
            
            hipDeviceSynchronize();



            // 3.1 Update global IPIV array and prepare indices for LASWP
            int *h_panel_ipiv = new int[panel_cols];
            hipMemcpy(h_panel_ipiv, d_IPIV_panel, panel_cols * sizeof(int), hipMemcpyDeviceToHost);

            // Convert local panel indices to global indices
            for (int j = 0; j < panel_cols; ++j) {
                // h_panel_ipiv[j] is 1-based local index within panel
                // Convert to global 1-based index for final IPIV output
                IPIV[k + j] = h_panel_ipiv[j] + k;
                // Also convert for LASWP kernel which expects global indices
                h_panel_ipiv[j] = h_panel_ipiv[j] + k;  
            }
            
            // Update the device array with global indices for LASWP
            hipMemcpy(d_IPIV_panel, h_panel_ipiv, panel_cols * sizeof(int), hipMemcpyHostToDevice);
            delete[] h_panel_ipiv;

            // 3.2 Apply permutations to FP64 matrix (kernel)
            LASWP_kernel << <grid_size(N), __threads_per_block__ >> > (d_A, N, k, panel_cols, d_IPIV_panel);
            hipDeviceSynchronize();


            // 4.1 Copy updated panel back for FP64 factorization
            // Copy updated panel from d_A back to d_P_FP64_NPV_buffer column by column
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    d_A + (k + col) * N + k,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }

            // 4.2 Panel LU factorization in FP64 without pivoting (kernel)
            int num_blocks_dgetf2 = grid_size(panel_rows);
            int threads_per_block_dgetf2 = __threads_per_block__;
            
            void* args_dgetf2[] = {&panel_rows, &panel_cols, &d_P_FP64_NPV_buffer, &panel_rows};
            
            hipError_t err_dgetf2 = hipLaunchCooperativeKernel((void*)dgetf2_native_npv, 
                                                              dim3(num_blocks_dgetf2), dim3(threads_per_block_dgetf2), 
                                                              args_dgetf2, 0, 0);
            if (err_dgetf2 != hipSuccess) {
                std::cout << "CUDA dgetf2 kernel error: " << hipGetErrorString(err_dgetf2) 
                         << " (panel_rows=" << panel_rows << ", num_blocks=" << num_blocks_dgetf2 << ")" << std::endl;
                return;
            }
            hipDeviceSynchronize();

            // 4.3 Copy back the panel to matrix A
            // Copy back the panel to matrix A column by column
            for (int col = 0; col < panel_cols; ++col) {
                hipMemcpy(
                    d_A + (k + col) * N + k,
                    d_P_FP64_NPV_buffer + col * panel_rows,
                    panel_rows * sizeof(double),
                    hipMemcpyDeviceToDevice
                );
            }
            // 5 Trailing submatrix update (cuBLAS)
            if (k + panel_cols < N) {
                int n = N - k - panel_cols;  // Number of columns in trailing matrix
                int m = N - k - panel_cols;  // Number of rows in trailing matrix
                
                // 5.1 Solve triangular system L21 * U12 = A12 (where A12 is the top-right block)
                // We need to solve L^-1 * A12 = U12, which is equivalent to L * U12 = A12
                // Since L is unit lower triangular, we use triangular solve
                double alpha = 1.0;
                hipblasDtrsm(
                    handle,
                    HIPBLAS_SIDE_LEFT,           // L is on the left
                    HIPBLAS_FILL_MODE_LOWER,     // L is lower triangular
                    HIPBLAS_OP_N,                // No transpose of L
                    HIPBLAS_DIAG_UNIT,           // Unit diagonal
                    panel_cols, n,              // dimensions: m=panel_cols, n=trailing_cols
                    &alpha,                     // alpha = 1.0
                    d_A + k * N + k, N,         // L11 (panel_cols x panel_cols)
                    d_A + (k + panel_cols) * N + k, N  // A12 -> U12 (panel_cols x n)
                );
                

                // 5.2 Update trailing submatrix A22 = A22 - L21 * U12
                alpha = -1.0; 
                double beta = 1.0;
                hipblasDgemm(
                    handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,                                      // no transpose L21, no transpose U12
                    m, n, panel_cols,                                              // dimensions: m, n, k
                    &alpha,                                                        // -1.0
                    d_A + k * N + k + panel_cols, N,                               // L21 (m x panel_cols)
                    d_A + (k + panel_cols) * N + k, N,                             // U12 (panel_cols x n)
                    &beta,                                                         // 1.0
                    d_A + (k + panel_cols) * N + k + panel_cols, N                 // A22 (m x n)
                );
            }
        }

    }

    // Copy matrix back to host
    hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_P_FP16_buffer);
    hipFree(d_P_FP64_NPV_buffer);
    hipFree(d_IPIV_panel);
    hipFree(d_IPIV);
}
